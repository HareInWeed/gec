#include "cuda_common.cuh"
#include <iomanip>

// hipError_t related

bool CudaErrorMatcher::match(hipError_t const &in) const {
    return in == expect_code;
}

std::string CudaErrorMatcher::describe() const {
    return std::string(hipGetErrorName(expect_code));
}

namespace Catch {

std::string StringMaker<hipError_t>::convert(hipError_t const &code) {
    return std::string() + "{" + hipGetErrorName(code) +
           "}: " + hipGetErrorString(code);
}

} // namespace Catch
